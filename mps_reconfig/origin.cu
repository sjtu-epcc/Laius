#include <cstdio>
#include <unistd.h>
#include <fstream>
#include <string>
#include <iostream>
#include <hip/hip_runtime.h>

#define cudachk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if(code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s:%d\n", hipGetErrorString(code), file, line);
    if(abort) exit(code);
  }
}

struct LargeArg {
  char data[512];
};

__global__ void kernel(size_t param1, float *param2, LargeArg param3, int nop) {
  __shared__ float data[128];
  float local = 1234.0;
  for(size_t w = 0; w < 10240; w++) {
    for(size_t i = 0; i < 128; i += 4) {
      local += data[i];
      local /= data[i+1];
      local -= data[i+2];
      local *= data[i+3];
    }
  }
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  idx %= 128;
  data[idx] += local;
}

int main() {
  //cudachk( cudaMalloc(&data, 128 * sizeof(float)) );
  //int self_pid = getpid();
  //printf("%d\n",self_pid);
  //std::ifstream self_map;
  //std::string self_map_file = "/proc/" + std::to_string(self_pid) + "/maps";
  //self_map.open(self_map_file, std::ios::in);
  //std::string str_line;
  //while(getline(self_map, str_line))
  //{
    //std::cout << str_line << std::endl;
  //}
  hipEvent_t local_start, local_stop;
  cudachk( hipEventCreate(&local_start) );
  cudachk( hipEventCreate(&local_stop) );
  size_t a = 1234;
  float *b;
  cudachk( hipMalloc(&b, 4) );
  LargeArg c;
  c.data[0] = 5;
  kernel<<<128, 128>>>(a, b ,c, 0);
  //kernel<<<1024, 1024>>>();
  const int iters = 1;
  for(int i = iters; i > 0; i--) {
    cudachk( hipEventRecord(local_start) );
    kernel<<<1024, 1024>>>(a, b, c, 0);
    cudachk( hipEventRecord(local_stop) );
    cudachk( hipEventSynchronize(local_stop) );
    float milliseconds = 0;
    cudachk( hipEventElapsedTime(&milliseconds, local_start, local_stop) );
    printf("origin %fms\n", milliseconds);
  }
  //cudaFree(data);
}

